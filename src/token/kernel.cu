
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <cuda.h>
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#include <vector>
#include <iostream>
#include <stdexcept>
#include "include/tokenise.hpp"

// Helper for CUDA Error Checking
#define CHECK_CUDA(call) do {   \
    cudaError_t err = call;     \
    if (err != cudaSuccess) {   \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", cudaGetErrorString(err), __FILE__, __LINE__);  \
        exit(EXIT_FAILURE);     \
    }                           \
} while (0)

// Basic XorShift32 PRNG for CUDA
// Each thread uses its unique thread index to get a distinct seed
__device__ unsigned int xorshift32_cuda(unsigned int x) {
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    return x;
}

// Function to convert a normalized [0,1] float to a custom range [r1, r2]
__device__ float scale_random_cuda(unsigned int* seed_ptr, float r1) {
    *seed_ptr = xorshift32_cuda(*seed_ptr);
    float normalized_val = (float)(*seed_ptr) / (float)UINT_MAX;
    return r1 + normalized_val * (10.0f - r1);
}

// CUDA Kernel to generate embeddings
__global__ void generate_embeddings_kernel(
    float* embeddings_out,
    int d_dim,
    float r1,
    float r2,
    unsigned int initial_seed_offset,
    int total_elements) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread ID

    if (idx < total_elements) {
        // Use global_id + an offset for a unique seed per thread
        unsigned int seed = initial_seed_offset + idx + 1; // +1 to avoid seed 0

        // Generate random float in [r1, r2]
        embeddings_out[idx] = scale_random_cuda(&seed, r1);
    }
}

/**
 * @brief Computes the inverse (v / ||v||^2) for a batch of vectors in parallel.
 * Each CUDA block is responsible for processing one vector (one row of the matrix).
 * @param output The output matrix (N x d), flattened.
 * @param input The input matrix (N x d), flattened.
 * @param N The number of vectors (rows).
 * @param d The dimension of each vector (columns).
 */
__global__ void batchedVectorInverseKernel(float* output, const float* input, int N, int d) 
{
    // Use dynamic shared memory, sized by the kernel launch.
    // This will hold the values for the reduction.
    extern __shared__ float s_data[];

    // Identify which row (vector) this block is working on.
    const int row_idx = blockIdx.y;

    // Identify the thread's index within the block and its global column index.
    const int tid_in_block = threadIdx.x;
    const int col_idx = blockIdx.x * blockDim.x + tid_in_block;

    // --- Step 1: Parallel Reduction to find the squared magnitude ---

    float my_val = 0.0f;
    // Load the thread's value from the input matrix if it's within bounds.
    if (col_idx < d) {
        my_val = input[row_idx * d + col_idx];
    }
    
    // Store the square of the value in shared memory for reduction.
    s_data[tid_in_block] = my_val * my_val;

    // Synchronize to make sure all threads have written their squared value to shared memory.
    __syncthreads();

    // Perform the reduction in shared memory.
    // Each thread adds its right-half neighbor's value to its own.
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid_in_block < s) {
            s_data[tid_in_block] += s_data[tid_in_block + s];
        }
        __syncthreads(); // Synchronize at each step of the reduction tree.
    }

    // After the loop, the first element of shared memory (s_data[0]) holds the
    // final squared magnitude for the entire row.
    const float squared_magnitude = s_data[0];

    // --- Step 2: Element-wise division ---

    // Ensure the thread is within bounds and the magnitude is non-zero to avoid division by zero.
    if (col_idx < d && squared_magnitude > 1e-9f) {
        output[row_idx * d + col_idx] = my_val / squared_magnitude;
    } else if (col_idx < d) {
        // Handle zero-magnitude vector case (output is all zeros).
        output[row_idx * d + col_idx] = 0.0f;
    }
}

// =================================================================================
// HOST-SIDE WRAPPER FUNCTIONS
// =================================================================================


// Host-side wrapper function
void tokeniser::cuEmbeddingFormula(std::vector<std::vector<float>>& embedding, const std::vector<float>& seeds_ignored, 
    int& d_dim, int& vocSize_val, float r1) 
{
    // Resize embedding vector to hold the results
    embedding.resize(vocSize_val, std::vector<float>(d_dim));

    size_t total_elements = (size_t)vocSize_val * d_dim;
    if (total_elements == 0) return;

    float* d_embeddings = nullptr; // Device pointer for embeddings

    // Allocate device memory
    CUDA_CHECK(cudaMalloc(&d_embeddings, total_elements * sizeof(float)));

    // Configure kernel launch parameters
    int threads_per_block = 256;
    int blocks_per_grid = (total_elements + threads_per_block - 1) / threads_per_block;

    // Use a time-based seed offset for better randomness across runs
    unsigned int initial_seed_offset = static_cast<unsigned int>(std::chrono::high_resolution_clock::now().time_since_epoch().count());

    // Launch kernel
    generate_embeddings_kernel<<<blocks_per_grid, threads_per_block>>>(
        d_embeddings,
        d_dim,
        r1,
        initial_seed_offset,
        total_elements
    );
    CUDA_CHECK(cudaGetLastError()); // Check for errors during kernel launch

    // Copy results back to host
    std::vector<float> flat_embeddings(total_elements);
    CUDA_CHECK(cudaMemcpy(flat_embeddings.data(), d_embeddings, total_elements * sizeof(float), cudaMemcpyDeviceToHost));

    // Copy flat_embeddings to the 2D embedding vector
    for (int i = 0; i < vocSize_val; ++i) {
        for (int j = 0; j < d_dim; ++j) {
            embedding[i][j] = flat_embeddings[i * d_dim + j];
        }
    }

    // Free device memory
    CUDA_CHECK(cudaFree(d_embeddings));
}


/**
 * @brief Host wrapper to calculate batched vector inverses on the GPU.
 * @param deEmbedding [out] 2D vector to store the results. Will be resized.
 * @param embedding [in] 2D vector of input vectors.
 * @param d [in] The dimension of each vector.
 * @param vocSize [in] The number of vectors.
 */
void tokeniser::cuVectorInverse(std::vector<std::vector<float>>& deEmbedding,
    const std::vector<std::vector<float>>& embedding, int& d, int& vocSize)
{
    if (vocSize == 0 || d == 0) return;
    if (embedding.size() != vocSize || embedding[0].size() != d) {
        throw std::runtime_error("Input embedding dimensions do not match vocSize and d.");
    }

    // 1. Resize output and flatten the 2D input vector for the GPU
    deEmbedding.assign(vocSize, std::vector<float>(d));
    std::vector<float> h_flat_input(vocSize * d);
    std::vector<float> h_flat_output(vocSize * d);
    for (int i = 0; i < vocSize; ++i) {
        for (int j = 0; j < d; ++j) {
            h_flat_input[i * d + j] = embedding[i][j];
        }
    }

    // 2. Allocate device memory
    float *d_input, *d_output;
    size_t total_size = (size_t)vocSize * d * sizeof(float);
    CHECK_CUDA(cudaMalloc(&d_input, total_size));
    CHECK_CUDA(cudaMalloc(&d_output, total_size));

    // 3. Copy flattened input data to device
    CHECK_CUDA(cudaMemcpy(d_input, h_flat_input.data(), total_size, cudaMemcpyHostToDevice));

    // 4. Configure and launch kernel
    const int block_size = 256; // Must be power of 2 for this reduction
    dim3 grid_dim((d + block_size - 1) / block_size, vocSize, 1);
    dim3 block_dim(block_size, 1, 1);
    size_t shared_mem_size = block_dim.x * sizeof(float);
    batchedVectorInverseKernel<<<grid_dim, block_dim, shared_mem_size>>>(d_output, d_input, vocSize, d);
    CHECK_CUDA(cudaGetLastError());

    // 5. Copy flat results back to host
    CHECK_CUDA(cudaMemcpy(h_flat_output.data(), d_output, total_size, cudaMemcpyDeviceToHost));

    // 6. "Un-flatten" the results into the 2D output vector
    for (int i = 0; i < vocSize; ++i) {
        for (int j = 0; j < d; ++j) {
            deEmbedding[i][j] = h_flat_output[i * d + j];
        }
    }

    // 7. Free device memory
    CHECK_CUDA(cudaFree(d_input));
    CHECK_CUDA(cudaFree(d_output));
}

#endif