
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <cuda.h>
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#include <vector>
#include <iostream>
#include <stdexcept>
#include "include/tokenise.hpp"

// Helper for CUDA Error Checking
#define CHECK_CUDA(call) do { \
    cudaError_t err = call; \
    if (err != cudaSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", cudaGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

/** 
 * --> f(i, j, seed) = (i * j + 1) * C * (seed^[j%d])
 * where: C = 0.01, x = seed, and  d is the embedding dimension.
 */
__global__ void embeddingFormulaBatchKernel(float* all_embeddings, const float* all_seeds, 
    const int N, const int d) 
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < d) {
        float seed = all_seeds[i];
        const float C = 0.01f;
        float result = static_cast<float>(i * j + 1) * C;
        int exponent = j;
        result *= powf(seed, static_cast<float>(exponent));
        all_embeddings[i * d + j] = result;
    }
}


/**
 * @brief Computes the inverse (v / ||v||^2) for a batch of vectors in parallel.
 * Each CUDA block is responsible for processing one vector (one row of the matrix).
 * @param output The output matrix (N x d), flattened.
 * @param input The input matrix (N x d), flattened.
 * @param N The number of vectors (rows).
 * @param d The dimension of each vector (columns).
 */
__global__ void batchedVectorInverseKernel(float* output, const float* input, int N, int d) {
    // Use dynamic shared memory, sized by the kernel launch.
    // This will hold the values for the reduction.
    extern __shared__ float s_data[];

    // Identify which row (vector) this block is working on.
    const int row_idx = blockIdx.y;

    // Identify the thread's index within the block and its global column index.
    const int tid_in_block = threadIdx.x;
    const int col_idx = blockIdx.x * blockDim.x + tid_in_block;

    // --- Step 1: Parallel Reduction to find the squared magnitude ---

    float my_val = 0.0f;
    // Load the thread's value from the input matrix if it's within bounds.
    if (col_idx < d) {
        my_val = input[row_idx * d + col_idx];
    }
    
    // Store the square of the value in shared memory for reduction.
    s_data[tid_in_block] = my_val * my_val;

    // Synchronize to make sure all threads have written their squared value to shared memory.
    __syncthreads();

    // Perform the reduction in shared memory.
    // Each thread adds its right-half neighbor's value to its own.
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid_in_block < s) {
            s_data[tid_in_block] += s_data[tid_in_block + s];
        }
        __syncthreads(); // Synchronize at each step of the reduction tree.
    }

    // After the loop, the first element of shared memory (s_data[0]) holds the
    // final squared magnitude for the entire row.
    const float squared_magnitude = s_data[0];

    // --- Step 2: Element-wise division ---

    // Ensure the thread is within bounds and the magnitude is non-zero to avoid division by zero.
    if (col_idx < d && squared_magnitude > 1e-9f) {
        output[row_idx * d + col_idx] = my_val / squared_magnitude;
    } else if (col_idx < d) {
        // Handle zero-magnitude vector case (output is all zeros).
        output[row_idx * d + col_idx] = 0.0f;
    }
}

// =================================================================================
// HOST-SIDE WRAPPER FUNCTIONS
// =================================================================================

/**
 * @brief Host wrapper to generate embeddings on the GPU.
 * @param embedding [out] 2D vector to store the results. Will be resized.
 * @param seeds [in] 1D vector of seed values, one for each token.
 * @param d [in] The embedding dimension.
 * @param vocSize [in] The number of tokens/seeds (N).
 */
void tokeniser::cuEmbeddingFormula(std::vector<std::vector<float>>& embedding,
    const std::vector<float>& seeds, int& d, int& vocSize) 
{
    if (vocSize == 0 || d == 0) return;
    if (seeds.size() != vocSize) {
        throw std::runtime_error("Seed vector size must match vocSize.");
    }

    // 1. Resize output vector and create a flat buffer for GPU results
    embedding.assign(vocSize, std::vector<float>(d));
    std::vector<float> h_flat_output(vocSize * d);
    
    // 2. Allocate device memory
    float *d_seeds, *d_embeddings;
    CHECK_CUDA(cudaMalloc(&d_seeds, vocSize * sizeof(float)));
    CHECK_CUDA(cudaMalloc(&d_embeddings, (size_t)vocSize * d * sizeof(float)));

    // 3. Copy input seeds to device
    CHECK_CUDA(cudaMemcpy(d_seeds, seeds.data(), vocSize * sizeof(float), cudaMemcpyHostToDevice));

    // 4. Configure and launch kernel
    dim3 block_dim(16, 16);
    dim3 grid_dim((d + block_dim.x - 1) / block_dim.x, (vocSize + block_dim.y - 1) / block_dim.y);
    embeddingFormulaBatchKernel<<<grid_dim, block_dim>>>(d_embeddings, d_seeds, vocSize, d);
    CHECK_CUDA(cudaGetLastError());

    // 5. Copy flat results back to host
    CHECK_CUDA(cudaMemcpy(h_flat_output.data(), d_embeddings, (size_t)vocSize * d * sizeof(float), cudaMemcpyDeviceToHost));
    
    // 6. "Un-flatten" the results into the 2D output vector
    for (int i = 0; i < vocSize; ++i) {
        for (int j = 0; j < d; ++j) {
            embedding[i][j] = h_flat_output[i * d + j];
        }
    }

    // 7. Free device memory
    CHECK_CUDA(cudaFree(d_seeds));
    CHECK_CUDA(cudaFree(d_embeddings));
}

/**
 * @brief Host wrapper to calculate batched vector inverses on the GPU.
 * @param deEmbedding [out] 2D vector to store the results. Will be resized.
 * @param embedding [in] 2D vector of input vectors.
 * @param d [in] The dimension of each vector.
 * @param vocSize [in] The number of vectors.
 */
void cuVectorInverse(std::vector<std::vector<float>>& deEmbedding,
    const std::vector<std::vector<float>>& embedding, int& d, int& vocSize)
{
    if (vocSize == 0 || d == 0) return;
    if (embedding.size() != vocSize || embedding[0].size() != d) {
        throw std::runtime_error("Input embedding dimensions do not match vocSize and d.");
    }

    // 1. Resize output and flatten the 2D input vector for the GPU
    deEmbedding.assign(vocSize, std::vector<float>(d));
    std::vector<float> h_flat_input(vocSize * d);
    std::vector<float> h_flat_output(vocSize * d);
    for (int i = 0; i < vocSize; ++i) {
        for (int j = 0; j < d; ++j) {
            h_flat_input[i * d + j] = embedding[i][j];
        }
    }

    // 2. Allocate device memory
    float *d_input, *d_output;
    size_t total_size = (size_t)vocSize * d * sizeof(float);
    CHECK_CUDA(cudaMalloc(&d_input, total_size));
    CHECK_CUDA(cudaMalloc(&d_output, total_size));

    // 3. Copy flattened input data to device
    CHECK_CUDA(cudaMemcpy(d_input, h_flat_input.data(), total_size, cudaMemcpyHostToDevice));

    // 4. Configure and launch kernel
    const int block_size = 256; // Must be power of 2 for this reduction
    dim3 grid_dim((d + block_size - 1) / block_size, vocSize, 1);
    dim3 block_dim(block_size, 1, 1);
    size_t shared_mem_size = block_dim.x * sizeof(float);
    batchedVectorInverseKernel<<<grid_dim, block_dim, shared_mem_size>>>(d_output, d_input, vocSize, d);
    CHECK_CUDA(cudaGetLastError());

    // 5. Copy flat results back to host
    CHECK_CUDA(cudaMemcpy(h_flat_output.data(), d_output, total_size, cudaMemcpyDeviceToHost));

    // 6. "Un-flatten" the results into the 2D output vector
    for (int i = 0; i < vocSize; ++i) {
        for (int j = 0; j < d; ++j) {
            deEmbedding[i][j] = h_flat_output[i * d + j];
        }
    }

    // 7. Free device memory
    CHECK_CUDA(cudaFree(d_input));
    CHECK_CUDA(cudaFree(d_output));
}

#endif